/*
============================================================================
Filename    : algorithm.c
Author      : Olivier Cloux
SCIPER      : 236079
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

#define CENTER_VAL 1000

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<(int) iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;

            }
        }
        output[(length/2-1)*length+(length/2-1)] = CENTER_VAL;
        output[(length/2)*length+(length/2-1)]   = CENTER_VAL;
        output[(length/2-1)*length+(length/2)]   = CENTER_VAL;
        output[(length/2)*length+(length/2)]     = CENTER_VAL;

        temp = input;
        input = output;
        output = temp;
    }
}

__global__ void heattransfer(double *input, double *output, int length)
{
  int i = blockIdx.x + 1; //shifting to not consider
  int j = threadIdx.x + 1;
  output[i*length + j] = (input[(i-1)*(length)+(j-1)] +
                      input[(i-1)*(length)+(j)]   +
                      input[(i-1)*(length)+(j+1)] +
                      input[(i)*(length)+(j-1)]   +
                      input[(i)*(length)+(j)]     +
                      input[(i)*(length)+(j+1)]   +
                      input[(i+1)*(length)+(j-1)] +
                      input[(i+1)*(length)+(j)]   +
                      input[(i+1)*(length)+(j+1)] ) / 9;

  output[(length/2-1)*length+(length/2-1)] = CENTER_VAL;
  output[(length/2)*length+(length/2-1)]   = CENTER_VAL;
  output[(length/2-1)*length+(length/2)]   = CENTER_VAL;
  output[(length/2)*length+(length/2)]     = CENTER_VAL;
}

__global__ void heattransfer2(double *input, double *output, int length, int casePerBlock)
{
  int i = blockIdx.x + 1; //shifting to not consider
  int j = threadIdx.x*casePerBlock + 1;
  for (size_t k = 0; k < casePerBlock; k++) {
      // j = max((j + k) % length-2, 1);

      output[i*length + j] = (input[(i-1)*(length)+(j-1)] +
                          input[(i-1)*(length)+(j)]   +
                          input[(i-1)*(length)+(j+1)] +
                          input[(i)*(length)+(j-1)]   +
                          input[(i)*(length)+(j)]     +
                          input[(i)*(length)+(j+1)]   +
                          input[(i+1)*(length)+(j-1)] +
                          input[(i+1)*(length)+(j)]   +
                          input[(i+1)*(length)+(j+1)] ) / 9;

      output[(length/2-1)*length+(length/2-1)] = CENTER_VAL;
      output[(length/2)*length+(length/2-1)]   = CENTER_VAL;
      output[(length/2-1)*length+(length/2)]   = CENTER_VAL;
      output[(length/2)*length+(length/2)]     = CENTER_VAL;
  }
}
// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
  //Cuda events for calculating elapsed time
  hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
  hipEventCreate(&cpy_H2D_start);
  hipEventCreate(&cpy_H2D_end);
  hipEventCreate(&cpy_D2H_start);
  hipEventCreate(&cpy_D2H_end);
  hipEventCreate(&comp_start);
  hipEventCreate(&comp_end);

  /* Preprocessing goes here */
  size_t size = length*length*sizeof(double);
  double* in;
  double* out;
  double* temp;
  int casePerBlock = ((length-2)*(length-2) + 512 - 1) / 512;
  int threadePerBlock = 0;
  if(length-2 > 512){
    threadePerBlock = 512;
  } else {
    threadePerBlock = length-2;
  }
  if(hipMalloc((void **) &in, size) != hipSuccess){
    return;
  }

  if(hipMalloc((void **) &out, size) != hipSuccess){
    hipFree(in);
    return;
  }

  hipEventRecord(cpy_H2D_start);
  /* Copying array from host to device goes here */
  if(hipMemcpy(in, input, size, hipMemcpyHostToDevice) != hipSuccess){
    hipFree(in);
    hipFree(out);
    return;
  };

  hipEventRecord(cpy_H2D_end);
  hipEventSynchronize(cpy_H2D_end);

  hipEventRecord(comp_start);
  /*GPU computation*/
  for (size_t k = 0; k < iterations; k++) {
    /*Starting a kernel with each iteration to ensure synchronization
    * between blocks*/
    heattransfer<<<length-2, length-2>>>(in, out, length);
    // heattransfer2<<<length-2, threadePerBlock>>>(in, out, length, casePerBlock);
    temp = in;
    in = out;
    out = temp;
  }
  hipEventRecord(comp_end);
  hipEventSynchronize(comp_end);

  hipEventRecord(cpy_D2H_start);
  /*copy result from device to host*/
  if(iterations%2 == 0){
    hipMemcpy(output, in, size, hipMemcpyDeviceToHost);
  } else{
    hipMemcpy(output, out, size, hipMemcpyDeviceToHost);
  }

  hipEventRecord(cpy_D2H_end);
  hipEventSynchronize(cpy_D2H_end);

  hipFree(in);
  hipFree(out);

  float time;
  hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
  cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

  hipEventElapsedTime(&time, comp_start, comp_end);
  cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

  hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
  cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}
