/*
============================================================================
Filename    : algorithm.c
Author      : Samuel Chassot and Simon Wicky
SCIPER      : 270955, 260589
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

#define INPUT(X,Y) input[(X) * length + (Y)]
#define OUTPUT(X,Y) output[(X) * length + (Y)] 

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<(int) iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;

            }
        }
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}

__global__ void array_process_GPU(double *input, double *output, int length){
    //double *temp;
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if(y > 0 && y < length-1 && x > 0 && x < length - 1 ){
        OUTPUT(x,y) = (INPUT(x-1,y-1) +
                            INPUT(x-1,y)   +
                            INPUT(x-1,y+1) +
                            INPUT(x,y-1)   +
                            INPUT(x,y)     +
                            INPUT(x,y+1)   +
                            INPUT(x+1,y-1) +
                            INPUT(x+1,y)   +
                            INPUT(x+1,y+1) ) / 9;
    }
    if ((x == length / 2 || x == length / 2 - 1) &&  (y == length / 2 || y == length / 2 - 1)){
    	OUTPUT(x,y) = 1000;
    }
    INPUT(x,y) = 1000;
    OUTPUT(x,y) = 1000;

    //temp = input;
    //input = output;
    //output = temp;
}


// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    //Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing goes here */
    double* input_GPU;
    double* output_GPU;
    int size = length * length;
    hipMalloc((void**) &input_GPU, size);
    hipMalloc((void**) &output_GPU, size);

    hipEventRecord(cpy_H2D_start);


    /* Copying array from host to device goes here */

    hipMemcpy(input_GPU, input, size, hipMemcpyHostToDevice);
    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    //Copy array from host to device


    hipEventRecord(comp_start);
    /* GPU calculation goes here */
    for(int n=0; n < iterations; n++) {
    	array_process_GPU <<<100,100>>> (input_GPU, output_GPU, length);
    }


    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);


    /* Copying array from device to host goes here */
    hipMemcpy(output_GPU, output, size, hipMemcpyDeviceToHost);

    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    /* Postprocessing goes here */



    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}