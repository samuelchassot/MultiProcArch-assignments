/*
============================================================================
Filename    : algorithm.c
Author      : Samuel Chassot and Simon Wicky
SCIPER      : 270955, 260589
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

#define INPUT(X,Y) input[(X) * length + (Y)]
#define OUTPUT(X,Y) output[(X) * length + (Y)] 

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<(int) iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;

            }
        }
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}

__global__ void array_process_GPU(double *input, double *output, int length){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if(y > 0 && y < length - 1 && x > 0 && x < length - 1 ){
        OUTPUT(x,y) = (INPUT(x-1,y-1) +
                            INPUT(x-1,y)   +
                            INPUT(x-1,y+1) +
                            INPUT(x,y-1)   +
                            INPUT(x,y)     +
                            INPUT(x,y+1)   +
                            INPUT(x+1,y-1) +
                            INPUT(x+1,y)   +
                            INPUT(x+1,y+1) ) / 9;
    }
    if ((x == length / 2 || x == length / 2 - 1) &&  (y == length / 2 || y == length / 2 - 1)){
    	OUTPUT(x,y) = 1000;
    }


}


// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    //Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing goes here */
    double* input_GPU;
    double* output_GPU;
    double* temp;
    int size = length * length * sizeof(double);

    if (hipMalloc((void**) &input_GPU, size) != hipSuccess){
        cout<<"Couldn't alloc mem for in";
        return;
    }
    if (hipMalloc((void**) &output_GPU, size) != hipSuccess){
        cout<<"Couldn't alloc mem for out";
        return;
    }

    hipEventRecord(cpy_H2D_start);


    /* Copying array from host to device goes here */

    hipMemcpy(input_GPU, input, size, hipMemcpyHostToDevice);

    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    //Copy array from host to device
    dim3 thrsPerBlock(10,10);
    dim3 nBlks(10,10);

    hipEventRecord(comp_start);
    /* GPU calculation goes here */
    for(int n=0; n < iterations; n++) {
    	array_process_GPU <<<nBlks,thrsPerBlock>>> (input_GPU, output_GPU, length);
        temp = input_GPU;
        input_GPU = output_GPU;
        output_GPU = temp;
    }


    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);


    /* Copying array from device to host goes here */
    if (iterations % 2 == 0) {
        hipMemcpy(output, input_GPU, size, hipMemcpyDeviceToHost);
    } else {
        hipMemcpy(output, output_GPU, size, hipMemcpyDeviceToHost);
    }

    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    /* Postprocessing goes here */



    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}